#include "hip/hip_runtime.h"
#include "myKernel.cuh"
#include <opencv2/core.hpp>
#include <opencv2/cudev.hpp>
#include <hip/hip_runtime.h>
#include <>

__global__ void myKernel(cv::cuda::PtrStepSz<uchar> src, cv::cuda::PtrStepSz<uchar> dst)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    if((x < dst.cols) && (y < dst.rows)){
        dst.ptr(y)[x] = UCHAR_MAX - src.ptr(y)[x];
    }
}

void launchMyKernel(cv::cuda::GpuMat& src, cv::cuda::GpuMat& dst)
{
    cv::cuda::PtrStepSz<uchar> pSrc = 
        cv::cuda::PtrStepSz<uchar>(src.rows, src.cols * src.channels(), src.ptr<uchar>(), src.step);
    cv::cuda::PtrStepSz<uchar> pDst = 
        cv::cuda::PtrStepSz<uchar>(dst.rows, dst.cols * dst.channels(), dst.ptr<uchar>(), dst.step);

    const dim3 block(64, 8);
    const dim3 grid(cv::cudev::divUp(src.cols, block.x), cv::cudev::divUp(src.rows, block.y));

    myKernel<<<grid, block>>>(pSrc, pDst);
}

